#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "heat_cuda.h"

#define N 10
#define DIM 500

__global__ void kernel( void )
{
	return;
}

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

void cuda_devices()
{
	int count, i;
	hipDeviceProp_t prop;

	hipGetDeviceCount(&count);
	for(i=0; i<count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("name=%s\n", prop.name);
		printf("mem (Mb)=%d\n", prop.totalGlobalMem/(1024*1024));
		printf("capability=%d.%d\n", prop.major, prop.minor);
		printf("clock (MHz)=%d\n", prop.clockRate / 1000);
		printf("proc count=%d\n", prop.multiProcessorCount);
	}
}

void cuda_add()
{
	int i;
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void**) &dev_a, N*sizeof(int));
	hipMalloc((void**) &dev_b, N*sizeof(int));
	hipMalloc((void**) &dev_c, N*sizeof(int));

	for (i=0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	add<<<N,1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	for (i=0; i<N; i++) {
		printf("%d %d %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex(float a, float b) : r(a), i(b) {}
	__device__ float magnitude2(void) { return r * r + i * i; }
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
};

__device__ int julia(int x, int y)
{
	int i;
	const float scale = 2.0;
	float jx = scale * (float) (DIM/2 - x) / (DIM/2);
	float jy = scale * (float) (DIM/2 - y) / (DIM/2);
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);
	for (i=0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}
	return 1;
}

__global__ void julia_kernel(unsigned char *ptr)
{
	int i = blockIdx.x;
	int j = blockIdx.y;
	int idx = i + j * gridDim.x;
	int juliaValue = julia(i, j);
	ptr[idx*3 + 0] = 255 * juliaValue;
	ptr[idx*3 + 1] = 0;
	ptr[idx*3 + 2] = 0;
}

int save_image(char *path, unsigned char *image, int width, int height)
{
	FILE *f = NULL;

    if (image == NULL || path == NULL)
    	return -1;

	if ((f = fopen(path, "wb")) == NULL) {
		char *msg;
		if (asprintf(&msg, "Failed to open %s", path) < 0)
				perror("Failed to open output file");
		else
				perror(msg);
		return -1;
	}

	fprintf(f, "P6\n%d %d\n%d\n", width, height, 255);
	fwrite(image, sizeof(char) * 3, width * height, f);
	fclose(f);
	return 0;
}

void julia_func()
{
	unsigned char *ptr = (unsigned char *) calloc(DIM*DIM*3, 1);

	unsigned char *dev_ptr;

	hipMalloc((void**)&dev_ptr, DIM*DIM*3);
	dim3 grid(DIM, DIM);
	julia_kernel<<<grid,1>>>(dev_ptr);
	hipMemcpy(ptr, dev_ptr, DIM*DIM*3, hipMemcpyDeviceToHost);
	save_image("julia.ppm", ptr, DIM, DIM);
	hipFree(dev_ptr);
}

void kernel_wrapper() {

}
